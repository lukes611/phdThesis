#include "hip/hip_runtime.h"
/*
	ll_lib = Luke Lincoln's CV Library

	Author: Luke Lincoln

	contents description: 
		Contains the GPU cuda algorithms I wrote

	depends on: null
*/

#include <iostream>
#include <map>
#include <vector>
#include <functional>
#include <string>
#include "hip/hip_runtime.h"
#include ""
#include <opencv2\core\core.hpp>
#include <stdio.h>

#include <hipfft/hipfft.h>
#include <stdio.h>
#include <assert.h>
#include "code/gpu/ExternGPUPrograms.h"

using namespace std;
using namespace cv;






#define _USE_MATH_DEFINES
#include <math.h>
// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions

class LCuda_Memory_Ptr
{
public:
	const static int NON_HOST = 0;
	const static int HOST = 1;
	LCuda_Memory_Ptr(int type = NON_HOST)
	{
		_type = type;
		_ptr = NULL;
	}
	LCuda_Memory_Ptr(const LCuda_Memory_Ptr & i)
	{
		_type = i._type;
		_ptr = i._ptr;
	}
	LCuda_Memory_Ptr & operator = (const LCuda_Memory_Ptr & i)
	{
		_type = i._type;
		_ptr = i._ptr;
		return *this;
	}
	template <class T>
	T * operator () ()
	{
		return (T*)_ptr;
	}
	template <class T>
	T * pointer()
	{
		return (T*)_ptr;
	}
	template <class T>
	bool new_(int _size)
	{
		hipError_t cudaStatus;

		if(_type == NON_HOST)
			cudaStatus = hipMalloc((void**)&_ptr, _size * sizeof(T));
		else
			cudaStatus = hipHostMalloc((void**)&_ptr, _size * sizeof(T));
		if (cudaStatus != hipSuccess)
		{
			return false;
		}
		return true;
	}
	template <class T>
	bool copy_into(int _size, T * _data)
	{
		hipError_t cudaStatus = hipMemcpy(_ptr, _data, _size * sizeof(T), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
		{
			return false;
		}
		return true;
	}
	template <class T>
	void new_(int _size, T * _data, bool & alloc_worked, bool & cpy_worked)
	{
		alloc_worked = false;
		cpy_worked = false;
		if(!new_<T>(_size)) return;
		alloc_worked = true;
		cpy_worked = copy_into<T>(_size, _data);
	}
	bool delete_()
	{
		if(_type == NON_HOST)
			hipFree(_ptr);
		else
			hipHostFree(_ptr);
		return true;
	}
	template <class T>
	bool retrieve(T * _data, int _size)
	{
		hipError_t cudaStatus = hipMemcpy(_data, _ptr, _size * sizeof(T), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess)
		{
			return false;
		}
		return true;
	}
	void make_type_host()
	{
		_type = HOST;
	}
	void make_type_non_host()
	{
		_type = NON_HOST;
	}
private:
	void * _ptr;
	int _type;
};

class LCuda_Host_Manager
{
public:
	LCuda_Host_Manager()
	{
		_named_ptrs = new map<string,LCuda_Memory_Ptr>;
	}
	~LCuda_Host_Manager()
	{
		delete_all();
		delete _named_ptrs;
	}
	LCuda_Host_Manager(const LCuda_Host_Manager & i)
	{
		shallow_copy(i);
	}
	LCuda_Host_Manager & operator = (const LCuda_Host_Manager & i)
	{
		if(this == &i) return *this;
		shallow_copy(i);
		return *this;
	}
	bool delete_(string name)
	{
		if(is_in(name))
		{
			bool rv = _named_ptrs->operator[](name).delete_();
			_named_ptrs->erase(name);
			return rv;
		}
		_latest_error = "could not free " + name;
		return false;
	}
	void * operator [] (string name)
	{
		if(is_in(name))
			return (_named_ptrs->operator[](name)).pointer<void>();
		_latest_error = "could not find: " + name + " when accessing.";
		return NULL;
	}
	template <class T>
	T * at(string name)
	{
		if(is_in(name))
			return (_named_ptrs->operator[](name)).pointer<T>();
		_latest_error = "could not find: " + name + " when accessing.";
		return NULL;
	}
	bool set_default_device()
	{
		hipError_t cudaStatus = hipSetDevice(0);
		if(cudaStatus != hipSuccess)
		{
			_latest_error = "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?";
			return false;
		}
		return true;
	}
	bool is_in(string name)
	{
		return _named_ptrs->count(name) > 0;
	}
	vector<string> keys()
	{
		vector<string> rv;
		for(auto i = _named_ptrs->begin(); i != _named_ptrs->end(); i++)
			rv.push_back(i->first);
		return rv;
	}
	void print_all_keys()
	{
		cout << "Available keys:" << endl;
		vector<string> k = keys();
		for(string s : k) cout << "\t" << s << endl;
		cout << "end." << endl;
	}
	void delete_all()
	{
		vector<string> _keys = keys();
		for(auto i = _keys.begin(); i != _keys.end(); i++)
			delete_(*i);
	}
	template <class T>
	bool new_(string name, int _size)
	{
		if(is_in(name))
		{
			_latest_error = "tried to alloc to " + name + " but " + name + " is already in use";
			return false;
		}
		LCuda_Memory_Ptr mem(LCuda_Memory_Ptr::NON_HOST);
		bool rv = mem.new_<T>(_size);
		if(!rv)
		{
			_latest_error = "cuda alloc failed for " + name;
			return false;
		}
		_named_ptrs->operator[](name) = mem;
		return true;
	}
	template <class T>
	bool hnew_(string name, int _size)
	{
		if(is_in(name))
		{
			_latest_error = "tried to alloc to " + name + " but " + name + " is already in use";
			return false;
		}
		LCuda_Memory_Ptr mem(LCuda_Memory_Ptr::HOST);
		bool rv = mem.new_<T>(_size);
		if(!rv)
		{
			_latest_error = "cuda alloc failed for " + name;
			return false;
		}
		_named_ptrs->operator[](name) = mem;
		return true;
	}
	template <class T>
	bool new_(string name, int _size, T * data)
	{
		if(is_in(name))
		{
			_latest_error = "tried to alloc to " + name + " but " + name + " is already in use";
			return false;
		}
		LCuda_Memory_Ptr mem(LCuda_Memory_Ptr::NON_HOST);
		bool rv = mem.new_<T>(_size);
		if(!rv)
		{
			_latest_error = "cuda alloc failed for " + name;
			return false;
		}
		_named_ptrs->operator[](name) = mem;

		rv = mem.copy_into<T>(_size, data);
		if(!rv)
		{
			_latest_error = "cuda memcpy failed for " + name;
			return false;
		}
		return true;
	}
	template <class T>
	bool hnew_(string name, int _size, T * data)
	{
		if(is_in(name))
		{
			_latest_error = "tried to alloc to " + name + " but " + name + " is already in use";
			return false;
		}
		LCuda_Memory_Ptr mem(LCuda_Memory_Ptr::HOST);
		bool rv = mem.new_<T>(_size);
		if(!rv)
		{
			_latest_error = "cuda alloc failed for " + name;
			return false;
		}
		_named_ptrs->operator[](name) = mem;

		rv = mem.copy_into<T>(_size, data);
		if(!rv)
		{
			_latest_error = "cuda memcpy failed for " + name;
			return false;
		}
		return true;
	}
bool sync()
	{
		hipError_t cudaStatus = hipDeviceSynchronize();
		if(cudaStatus != hipSuccess)
		{
			_latest_error = string("hipDeviceSynchronize returned error code ") + std::to_string(cudaStatus) +  "after launching addKernel!\n";
			return false;
		}
		return true;
	}
	string error()
	{
		return _latest_error;
	}
	template <class T>
	bool collect(string name, int _size, T * data)
	{
		if(!is_in(name))
		{
			_latest_error = "could not find " + name + " to collect."; 
			return false;
		}
		bool rv = _named_ptrs->operator[](name).retrieve<T>(data, _size);
		if(!rv) _latest_error = "could not memcpy from " + name + " to your pointer in collect().";
		return rv;
	}
	template <class T>
	bool upload(string name, int _size, T * data)
	{
		if(!is_in(name))
		{
			_latest_error = "tried to upload to " + name + " but " + name + " is not available";
			return false;
		}
		if(!_named_ptrs->operator[](name).copy_into<T>(_size, data))
		{
			_latest_error = "cuda memcpy failed in upload for " + name;
			return false;
		}
		return true;
	}
private:
	void shallow_copy(const LCuda_Host_Manager & i)
	{
		_named_ptrs = i._named_ptrs;
	}
	map<string, LCuda_Memory_Ptr> * _named_ptrs;
	string _latest_error;
};

class VMatCufftComplex
{
public:
	hipfftComplex * d;
	int s, s2, s3;
	VMatCufftComplex()
	{
		s = s2 = s3 = 1;
		d = new hipfftComplex[s3];
	}
	VMatCufftComplex(int s)
	{
		this->s = s;
		s2 = s*s;
		s3 = s2*s;
		d = new hipfftComplex[s3];
	}
	VMatCufftComplex(const VMatCufftComplex & v)
	{
		this->s = v.s;
		s2 = s*s;
		s3 = s2*s;
		d = new hipfftComplex[s3];
		for(int i = 0; i < s3; i++) d[i] = v.d[i];
	}
	VMatCufftComplex(const VMat & v)
	{
		this->s = v.s;
		s2 = s*s;
		s3 = s2*s;
		d = new hipfftComplex[s3];
		for(int i = 0; i < s3; i++)
		{
			d[i].x = v.data[i];
			d[i].y = 0.0f;
		}
	}
	VMatCufftComplex(const VMat & re, const VMat & im)
	{
		this->s = re.s;
		s2 = s*s;
		s3 = s2*s;
		d = new hipfftComplex[s3];
		for(int i = 0; i < s3; i++)
		{
			d[i].x = re.data[i];
			d[i].y = im.data[i];
		}
	}
	VMatCufftComplex & operator = (const VMatCufftComplex & v)
	{
		if(this == &v) return *this;
		delete [] d;
		this->s = v.s;
		s2 = s*s;
		s3 = s2*s;
		d = new hipfftComplex[s3];
		for(int i = 0; i < s3; i++) d[i] = v.d[i];
		return *this;
	}
	~VMatCufftComplex()
	{
		delete [] d;
	}
	hipfftComplex & operator [] (int index)
	{
		return d[index];
	}
	hipfftComplex & operator () (int x, int y, int z)
	{
		return d[z*s2 + y*s + x];
	}
	VMat real()
	{
		VMat rv = s;
		for(int i = 0; i < rv.s3; i++) rv.data[i] = this->d[i].x;
		return rv;
	}
	VMat imag()
	{
		VMat rv = s;
		for(int i = 0; i < rv.s3; i++) rv.data[i] = this->d[i].y;
		return rv;
	}
	Point3i peak_real()
	{
		float peak = (*this)(0,0,0).x;
		Point3i rv(0,0,0);
		for(int z = 0; z < s; z++)
		{
			for(int y = 0; y < s; y++)
			{
				for(int x = 0; x < s; x++)
				{
					float v = (*this)(x,y,z).x;
					if(v > peak)
					{
						peak = v;
						rv.x = x; rv.y = y; rv.z = z;
					}
				}
			}
		}
		return rv;
	}
	static Point3i filter_phase_peak(Point3i a, int s)
	{
		function<int(int,int)> f = [](int a, int s) -> int { return (a > s/2) ? s-a: -a; };
		return Point3i(f(a.x,s), f(a.y,s), f(a.z,s));
	}
	static void phase_correlate_rst_adjust_rs(Point3i pc, float & rotation, float & scale, int s)
	{
		R3 q((float)pc.x, (float)pc.y, (float)pc.z);
		q.x *= (-360.0f / (float)s);
		q.y *= (180.0f / (float)s);
		q.z /= (((float) s) / log(((float) s) / 2.56f));
		q.z = exp(q.z);
		rotation = q.x;
		scale = 1.0f /  q.z;
	}
};


struct R3_
{
	float x, y ,z;
	__host__ __device__ void newR3_(float a, float b, float c)
	{
		x = a;
		y = b;
		z = c;
	}
	__host__ __device__ static void GetUnitPointFromAngle(float angle, float & x, float & y)
	{
		angle /= 57.2957795f;
		x = cos(angle);
		y = sin(angle);
	}
	__host__ __device__ void set_from_dual_angles(float a1, float a2)
	{
		if(a2 == 0.0f)
		{
			x = 0.0f;
			y = 1.0f;
			z = 0.0f;
			return;
		}else if(a2 == 180.0f)
		{
			x = 0.0f;
			y = -1.0f;
			z = 0.0f;
			return;
		}
		float x1, y1, x2, y2;
		float zdirx=0.0f, zdiry=0.0f, zdirz=1.0f;
		float ydirx=0.0f, ydiry=1.0f, ydirz=0.0f;
		float xdirx=1.0f, xdiry=0.0f, xdirz=0.0f;
		GetUnitPointFromAngle(a1, x1, y1);
		GetUnitPointFromAngle(a2, x2, y2);
		zdirx *= y1; zdiry *= y1; zdirz *= y1;
		xdirx *= x1; xdiry *= x1; xdirz *= x1;
		xdirx += zdirx; xdiry += zdiry; xdirz += zdirz; 

		xdirx *= y2; xdiry *= y2; xdirz *= y2;
		ydirx *= x2; ydiry *= x2; ydirz *= x2;
		xdirx += ydirx; xdiry += ydiry; xdirz += ydirz;
		x = xdirx;
		y = xdiry;
		z = xdirz;
	}
	__host__ __device__ void log_polar_inv(int s)
	{
		float sf = (float) s;
		x /= sf;
		x *= 360.0f;
		y /= sf;
		y *= 180.0f;
		float M = ((float) s) / log(((float) s) / 2.56f);
		z /= M;
		M = exp(z);
		float a1 = x;
		float a2 = y;
		set_from_dual_angles(a1, a2);
		x *= M;
		y *= M;
		z *= M;
		float hw = sf * 0.5f;
		x += hw;
		y += hw;
		z += hw;
	}

	__host__ __device__ void logonly_inv(int s)
	{
		float sf = (float)s;
		float sfh = sf * 0.5f;
		R3_ hw; hw.newR3_(sfh, sfh, sfh);
		float M = sf / log(sf / 2.56f);

		x -= hw.x;
		y -= hw.y;
		z -= hw.z;

		float mag = sqrt(x*x+y*y+z*z);
		
		x/=mag;
		y/=mag;
		z/=mag;

		mag /= M;
		mag = exp(mag);
		

		x*=mag;
		y*=mag;
		z*=mag;

		x+=hw.x;
		y+=hw.y;
		z+=hw.z;

	}

};

struct LMat_
{
	float * d;
	int r, c;
	__host__ __device__ void new_LMat_(int r, int c, float * d)
	{
		this->r = r;
		this->c = c;
		this->d = d;
	}
	__host__ __device__ float & at(int r, int c)
	{
		return this->d[r*this->c + c];
	}
	__host__ __device__ struct R3_ multiply(struct R3_ v)
	{
		struct R3_ rv;
		rv.newR3_(
			v.x*d[0] + v.y*d[1] + v.z*d[2] + d[3],
			v.x*d[4] + v.y*d[5] + v.z*d[6] + d[7],
			v.x*d[8] + v.y*d[9] + v.z*d[10] + d[11]
		);
		return rv;
	}
};

struct VMat_
{
	int s;
	float * d;
	__host__ __device__ void new_VMat_(int s, float * d)
	{
		this->s = s;
		this->d = d;
	}
	__host__ __device__ float & at(int x, int y, int z)
	{
		return d[z*s*s + y*s + x];
	}
	__host__ __device__ bool inbounds(int x, int y, int z)
	{
		return x>=0 && y>=0 && z>=0 && x<s && y<s && z<s;
	}
	__host__ __device__ float at(struct R3_ r)
	{
		float pix1, pix2, tmp, tmp2;
		int p1x = (int)r.x;
		int p1y = (int)r.y;
		int p1z = (int)r.z;
		
		float a = (inbounds(p1x,p1y, p1z))? (float) at(p1x, p1y, p1z): 0.0f;
		float b = (inbounds(p1x+1,p1y, p1z))? (float) at(p1x+1, p1y, p1z): 0.0f;
		float c = (inbounds(p1x, p1y+1, p1z))? (float) at(p1x, p1y+1, p1z): 0.0f;
		float d = (inbounds(p1x+1,p1y+1, p1z))? (float) at(p1x+1, p1y+1, p1z): 0.0f;
		p1z++;
		float e = (inbounds(p1x,p1y, p1z))? (float) at(p1x, p1y, p1z): 0.0f;
		float f = (inbounds(p1x+1,p1y, p1z))? (float) at(p1x+1, p1y, p1z): 0.0f;
		float g = (inbounds(p1x,p1y+1, p1z))? (float) at(p1x, p1y+1, p1z): 0.0f;
		float h = (inbounds(p1x+1,p1y+1, p1z))? (float) at(p1x+1, p1y+1, p1z): 0.0f;
		p1z--;
		float dx = r.x - (float) p1x;
		float dy = r.y - (float) p1y;
		float dz = r.z - (float) p1z;

		tmp = (1.0f-dx)*a + dx*b;
		tmp2 = (1.0f-dx)*c + dx*d;
		pix1 = (1.0f-dy)*tmp + dy*tmp2;

		tmp = (1.0f-dx)*e + dx*f;
		tmp2 = (1.0f-dx)*g + dx*h;
		pix2 = (1.0f-dy)*tmp + dy*tmp2;

		return (1.0f-dz)*pix1 + dz*pix2;
	}
};

struct R3_ newR3_(float a, float b, float c)
{
	struct R3_ rv;
	rv.x = a;
	rv.y = b;
	rv.z = c;
	return rv;
}

__global__ void multiply_spectrums(hipfftComplex * signal1, hipfftComplex * signal2, int N)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
	int z = blockIdx.z*blockDim.z + threadIdx.z;
	if(x >= N || y >= N || z >= N) return;

	int bid = z*N*N + y*N + x;

	signal2[bid].y = -signal2[bid].y;

	hipfftComplex tmp;
	tmp.x = signal1[bid].x*signal2[bid].x - signal1[bid].y*signal2[bid].y;
	tmp.y = signal1[bid].y*signal2[bid].x + signal1[bid].x*signal2[bid].y;


	float mag = sqrt(tmp.x*tmp.x + tmp.y*tmp.y);
	
	signal1[bid].x = tmp.x / mag;
	signal1[bid].y = tmp.y / mag;

}

__global__ void gpu_get_magnitude(hipfftComplex * signal1, float * output, int N)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
	int z = blockIdx.z*blockDim.z + threadIdx.z;
	if(x >= N || y >= N || z >= N) return;

	int bid = z*N*N + y*N + x;
	hipfftComplex tmp = signal1[bid];
	float mag = sqrt(tmp.x*tmp.x + tmp.y*tmp.y);
	output[bid] = mag;
}

__global__ void gpu_get_magnitude_swapQuads(hipfftComplex * signal1, float * output, int N)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
	int z = blockIdx.z*blockDim.z + threadIdx.z;
	int hw = N/2;
	if(x > N || y > N || z >= hw) return;

	int ox=x, oy=y, oz=z+hw;

	
	if(x<hw && y<hw)
	{
		ox += hw;
		oy += hw;
	}else if(x>=hw && y<hw)
	{
		ox -= hw;
		oy += hw;
	}else if(x<hw && y>=hw)
	{
		ox += hw;
		oy -= hw;
	}else if(x>=hw && y>=hw)
	{
		ox -= hw;
		oy -= hw;
	}

	int bid = z*N*N + y*N + x;
	int bid2 = oz*N*N + oy*N + ox;
	hipfftComplex tmp = signal1[bid];
	hipfftComplex tmp2 = signal1[bid2];
	float mag = sqrt(tmp.x*tmp.x + tmp.y*tmp.y);
	float mag2 = sqrt(tmp2.x*tmp2.x + tmp2.y*tmp2.y);

	output[bid] = mag2;
	output[bid2] = mag;
}

__global__ void copytoCufftComplex(hipfftComplex * signal1, float * input, int N)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
	int z = blockIdx.z*blockDim.z + threadIdx.z;
	if(x >= N || y >= N || z >= N) return;

	int bid = z*N*N + y*N + x;
	signal1[bid].x = input[bid];
	signal1[bid].y = 0.0f;
}

__global__ void laplacian_gpu(hipfftComplex * input, float * output, int N)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
	int z = blockIdx.z*blockDim.z + threadIdx.z;
	if(x >= N || y >= N || z >= N) return;

	int bid = z*N*N + y*N + x;

	if(x < 1 || y < 1 || z < 1 || x >= (N-1) || y >= (N-1) || z >= (N-1))
	{
		output[bid] = 0.0f;
		return;
	}

	//indexes 0 to 25
	float val = input[(z)*N*N + (y)*N + (x)].x * 26.0f;
	val += input[(z)*N*N + (y)*N + (x+1)].x * -1.0f;
	val += input[(z)*N*N + (y-1)*N + (x+1)].x * -1.0f;
	val += input[(z)*N*N + (y-1)*N + (x)].x * -1.0f;
	val += input[(z)*N*N + (y-1)*N + (x-1)].x * -1.0f;
	val += input[(z)*N*N + (y)*N + (x-1)].x * -1.0f;
	val += input[(z)*N*N + (y+1)*N + (x-1)].x * -1.0f;
	val += input[(z)*N*N + (y+1)*N + (x)].x * -1.0f;
	val += input[(z)*N*N + (y+1)*N + (x+1)].x * -1.0f;

	val += input[(z-1)*N*N + (y)*N + (x+1)].x * -1.0f;
	val += input[(z-1)*N*N + (y-1)*N + (x+1)].x * -1.0f;
	val += input[(z-1)*N*N + (y-1)*N + (x)].x * -1.0f;
	val += input[(z-1)*N*N + (y-1)*N + (x-1)].x * -1.0f;
	val += input[(z-1)*N*N + (y)*N + (x-1)].x * -1.0f;
	val += input[(z-1)*N*N + (y+1)*N + (x-1)].x * -1.0f;
	val += input[(z-1)*N*N + (y+1)*N + (x)].x * -1.0f;
	val += input[(z-1)*N*N + (y+1)*N + (x+1)].x * -1.0f;
	val += input[(z-1)*N*N + (y)*N + (x)].x * -1.0f;

	val += input[(z+1)*N*N + (y)*N + (x+1)].x * -1.0f;
	val += input[(z+1)*N*N + (y-1)*N + (x+1)].x * -1.0f;
	val += input[(z+1)*N*N + (y-1)*N + (x)].x * -1.0f;
	val += input[(z+1)*N*N + (y-1)*N + (x-1)].x * -1.0f;
	val += input[(z+1)*N*N + (y)*N + (x-1)].x * -1.0f;
	val += input[(z+1)*N*N + (y+1)*N + (x-1)].x * -1.0f;
	val += input[(z+1)*N*N + (y+1)*N + (x)].x * -1.0f;
	val += input[(z+1)*N*N + (y+1)*N + (x+1)].x * -1.0f;
	val += input[(z+1)*N*N + (y)*N + (x)].x * -1.0f;

	
	output[bid] = val;
}

//gpu transform volume input by matrix m, put result into volume output
__global__ void volume_transform(float * input, float * output, float * m, int N)
{
	
	int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
	int z = blockIdx.z*blockDim.z + threadIdx.z;
	if(x >= N || y >= N || z >= N) return;

	//create p
	struct R3_ p;
	p.newR3_( (float) x, (float) y, (float) z);

	//create matrix
	struct LMat_ mat;
	mat.new_LMat_(4, 4, m);

	//transform p
	p = mat.multiply(p);

	//creat vmats:
	VMat_ out; out.new_VMat_(N, output);
	VMat_ inp; inp.new_VMat_(N, input);

	//linearly interpolate from input to output according to p
	out.at(x,y,z) = inp.at(p); 
}

//re-written
__global__ void logpolar3d_gpu(float * input, float * output, int N)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
	int z = blockIdx.z*blockDim.z + threadIdx.z;
	if(x >= N || y >= N || z >= N) return;
	struct R3_ p; p.newR3_((float)x, (float)y, (float)z);
	p.log_polar_inv(N);

	struct VMat_ in; in.new_VMat_(N, input);
	struct VMat_ out; out.new_VMat_(N, output);

	
	out.at(x,y,z) = in.at(p);
}

__global__ void logonly3d_gpu(float * input, float * output, int N)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
	int z = blockIdx.z*blockDim.z + threadIdx.z;
	if(x >= N || y >= N || z >= N) return;
	struct R3_ p; p.newR3_((float)x, (float)y, (float)z);
	p.logonly_inv(N);

	struct VMat_ in; in.new_VMat_(N, input);
	struct VMat_ out; out.new_VMat_(N, output);

	
	out.at(x,y,z) = in.at(p);
}

//re-written
__global__ void logpolar3d_gpu_complex_out(float * input, hipfftComplex * output, int N)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
	int z = blockIdx.z*blockDim.z + threadIdx.z;
	if(x >= N || y >= N || z >= N) return;
	struct R3_ p; p.newR3_((float)x, (float)y, (float)z);
	p.log_polar_inv(N);

	VMat_ in; in.new_VMat_(N, input);

	output[z*N*N + y*N + x].x = in.at(p);
	output[z*N*N + y*N + x].y = 0.0f;
}

__global__ void hanning_gpu(hipfftComplex * input, int N)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
	int z = blockIdx.z*blockDim.z + threadIdx.z;
	if(x >= N || y >= N || z >= N) return;
	
	int hw = N / 2;
	float hw_dist = sqrt((float)(hw * hw));
	float dist = sqrt((float)((x-hw)*(x-hw) + (y-hw)*(y-hw) + (z-hw)*(z-hw)));
	dist = hw_dist - dist;
	hw_dist *= 2.0f;

	

	input[z*N*N + y*N + x].x *= (0.5f * (1.0f - cos((2.0f * M_PI * dist) / (hw_dist - 1.0f))));
	
}

__global__ void log_on_gpu(float * input, int N)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
	int z = blockIdx.z*blockDim.z + threadIdx.z;
	if(x >= N || y >= N || z >= N) return;
	input[z*N*N + y*N + x] = log(input[z*N*N + y*N + x]);
}



//vol1,vol2 were the volumes, N can be v1.s, returns true/false, and float rotation, float scale, and R3 translation
//xp,yp,zp were trans params
bool LLGPU_phase_correlate_rst(VMat vol1, VMat vol2, float & rotation, float & scale, Point3i & translation, bool hanning_window_on)
{
	LCuda_Host_Manager m;
	
	try
	{
		int S = vol1.s3;
		VMatCufftComplex vol1_ = vol1;
		VMatCufftComplex vol2_ = vol2;
		
		if(!m.new_<hipfftComplex>("vol1_", S, vol1_.d)) throw m.error();
		if(!m.new_<hipfftComplex>("vol2_", S, vol2_.d)) throw m.error();
		if(!m.new_<float>("tmp1", S)) throw m.error();
		if(!m.new_<float>("tmp2", S)) throw m.error();
		if(!m.new_<float>("matrix", 16)) throw m.error();


		//setup different types of GPU runs
		dim3 threadsPerBlock(8, 8, 8);
		dim3 numBlocks(vol1.s / threadsPerBlock.x, vol1.s / threadsPerBlock.y, vol1.s / threadsPerBlock.z);
		int hw = vol1.s/2;
		dim3 numBlocks4M(vol1.s / threadsPerBlock.x, vol1.s / threadsPerBlock.y, hw / threadsPerBlock.z);
		
		//hanning window
		if(hanning_window_on)
		{
			//hanning window
			hanning_gpu<<<numBlocks, threadsPerBlock>>>(m.at<hipfftComplex>("vol1_"), vol1.s);
			hanning_gpu<<<numBlocks, threadsPerBlock>>>(m.at<hipfftComplex>("vol2_"), vol1.s);
			if(!m.sync()) throw m.error();
		}

		//setup fft plan
		hipfftHandle plan;
		hipfftPlan3d(&plan, vol1.s, vol1.s, vol1.s, HIPFFT_C2C);

		//do fft on vol1_ and vol2_
		hipfftExecC2C(plan, m.at<hipfftComplex>("vol1_"), m.at<hipfftComplex>("vol1_"), HIPFFT_FORWARD);
		hipfftExecC2C(plan, m.at<hipfftComplex>("vol2_"), m.at<hipfftComplex>("vol2_"), HIPFFT_FORWARD);
		if(!m.sync()) throw m.error();
		
		//get the magnitude of both
		gpu_get_magnitude_swapQuads<<<numBlocks4M, threadsPerBlock>>>(m.at<hipfftComplex>("vol1_"), m.at<float>("tmp1"), vol1.s);
		gpu_get_magnitude_swapQuads<<<numBlocks4M, threadsPerBlock>>>(m.at<hipfftComplex>("vol2_"), m.at<float>("tmp2"), vol1.s);
		if(!m.sync()) throw m.error();

		//get the log of both
		log_on_gpu<<<numBlocks, threadsPerBlock>>>(m.at<float>("tmp1"), vol1.s);
		log_on_gpu<<<numBlocks, threadsPerBlock>>>(m.at<float>("tmp2"), vol1.s);
		if(!m.sync()) throw m.error();
	
		//get the log polar of tmp1 and tmp2 as tmp3 and tmp4 respectfully
		logpolar3d_gpu_complex_out<<<numBlocks, threadsPerBlock>>>(m.at<float>("tmp1"), m.at<hipfftComplex>("vol1_"), vol1.s);
		logpolar3d_gpu_complex_out<<<numBlocks, threadsPerBlock>>>(m.at<float>("tmp2"), m.at<hipfftComplex>("vol2_"), vol1.s);
		if(!m.sync()) throw m.error();
	

		//gpu_only_pc
		function<Point3i(LCuda_Host_Manager*,hipfftHandle*,string,string,VMatCufftComplex*)> f = 
		[](LCuda_Host_Manager * m, hipfftHandle * plan, string data1, string data2, VMatCufftComplex * cpu_a) -> Point3i
		{
			int S = cpu_a->s3;
			hipfftExecC2C(*plan, m->at<hipfftComplex>(data1), m->at<hipfftComplex>(data1), HIPFFT_FORWARD);
			hipfftExecC2C(*plan, m->at<hipfftComplex>(data2), m->at<hipfftComplex>(data2), HIPFFT_FORWARD);
			if(!m->sync()) throw m->error();

			dim3 threadsPerBlock(8, 8, 8);
			dim3 numBlocks(cpu_a->s / threadsPerBlock.x, cpu_a->s / threadsPerBlock.y, cpu_a->s / threadsPerBlock.z);
			multiply_spectrums<<<numBlocks, threadsPerBlock>>>(m->at<hipfftComplex>(data1), m->at<hipfftComplex>(data2), cpu_a->s);
			if(!m->sync()) throw m->error();

			hipfftExecC2C(*plan, m->at<hipfftComplex>(data1), m->at<hipfftComplex>(data1), HIPFFT_BACKWARD);

			if(!m->collect<hipfftComplex>(data1, S, cpu_a->d)) throw m->error();
			return cpu_a->filter_phase_peak(cpu_a->peak_real(), cpu_a->s);
		};

		//end here

		//Phase Correlate
		translation = f(&m, &plan, "vol1_", "vol2_", &vol1_);
		if(!m.sync()) throw m.error();
		VMatCufftComplex::phase_correlate_rst_adjust_rs(translation, rotation, scale, vol1.s);
		
		//copy from vol1 to tmp1, and from vol2_ to "vol2_"
		if(!m.upload<float>("tmp1", S, vol1.data)) throw m.error();
		if(!m.upload<hipfftComplex>("vol2_", S, vol2_.d)) throw m.error();
		
		//transform tmp1 by R/S and set into tmp2
		Mat transformation_matrix = VMat::transformation_matrix(vol1.s, 0.0f, rotation, 0.0f, scale, 0.0f, 0.0f, 0.0f);
		transformation_matrix = transformation_matrix.inv();
		if(!m.upload<float>("matrix", 16, (float*)transformation_matrix.data)) throw m.error();
		volume_transform<<<numBlocks, threadsPerBlock>>>(m.at<float>("tmp1"), m.at<float>("tmp2"), m.at<float>("matrix"), vol1.s);
		if(!m.sync()) throw m.error();

		//copy from tmp2 to vol1_
		//dim3 threadsPerBlock11(9, 9, 9);
		copytoCufftComplex<<<numBlocks, threadsPerBlock>>>(m.at<hipfftComplex>("vol1_"), m.at<float>("tmp2"), vol1.s);//wastmp2
		if(!m.sync()) throw m.error();
	
		if(!m.upload<hipfftComplex>("vol2_", S, vol2_.d)) throw m.error();
	
		//final PC
		translation = f(&m, &plan, "vol1_", "vol2_", &vol1_);
		if(!m.sync()) throw m.error();
	
		//destroy plan
		hipfftDestroy(plan);
	}catch(string e)
	{
		cout << "error on LLGPU_phase_correlate_rst() -> " << e << endl;
		m.print_all_keys();
		return false;
	}
	return true;
}

bool LLGPU_phase_correlate(VMat & v1, VMat & v2, Point3i & rv)
{
	LCuda_Host_Manager m;
	try
	{
		int S = v1.s3;
		VMatCufftComplex v1_ = v1;
		VMatCufftComplex v2_ = v2;
		
		//setup the fft plan
		hipfftHandle plan;
		hipfftPlan3d(&plan, v1.s, v1.s, v1.s, HIPFFT_C2C);

		//setup gpu data
		if(!m.new_<hipfftComplex>("v1_", S, v1_.d)) throw m.error();
		if(!m.new_<hipfftComplex>("v2_", S, v2_.d)) throw m.error();
		
		//do PC
		hipfftExecC2C(plan, m.at<hipfftComplex>("v1_"), m.at<hipfftComplex>("v1_"), HIPFFT_FORWARD);
		hipfftExecC2C(plan, m.at<hipfftComplex>("v2_"), m.at<hipfftComplex>("v2_"), HIPFFT_FORWARD);
		if(!m.sync()) throw m.error();
		dim3 threadsPerBlock(8, 8, 8);
		dim3 numBlocks(v1.s / threadsPerBlock.x, v1.s / threadsPerBlock.y, v1.s / threadsPerBlock.z);
		multiply_spectrums<<<numBlocks, threadsPerBlock>>>(m.at<hipfftComplex>("v1_"), m.at<hipfftComplex>("v2_"), v1.s);
		if(!m.sync()) throw m.error();

		hipfftExecC2C(plan, m.at<hipfftComplex>("v1_"), m.at<hipfftComplex>("v1_"), HIPFFT_BACKWARD);
		if(!m.sync()) throw m.error();
		if(!m.collect<hipfftComplex>("v1_", S, v1_.d)) throw m.error();
		rv = v1_.peak_real();
		
		//destroy the plan
		hipfftDestroy(plan);
	}catch(string e)
	{
		cout << "error in gpu_phase_correlate() -> " << e << endl;
		return false;
	}
	return true;
}

bool LLGPU_transform(VMat & v, R3 rotation, float scale, R3 translation)
{
	int S = v.s3;
	Mat transformation_matrix = VMat::transformation_matrix(v.s, rotation.x,rotation.y,rotation.z,scale,translation.x,translation.y,translation.z);
	transformation_matrix = transformation_matrix.inv();
	LCuda_Host_Manager m;
	try
	{
		if(!m.set_default_device()) throw m.error();
		if(!m.new_<float>("input", S, v.data)) throw m.error();
		if(!m.new_<float>("output", S)) throw m.error();
		if(!m.new_<float>("matrix", 16, (float*)transformation_matrix.data)) throw m.error();
		dim3 threadsPerBlock(8, 8, 8);
		dim3 numBlocks(v.s / threadsPerBlock.x, v.s / threadsPerBlock.y, v.s / threadsPerBlock.z);
		volume_transform<<<numBlocks, threadsPerBlock>>>(m.at<float>("input"), m.at<float>("output"), m.at<float>("matrix"), v.s);
		auto cudaStatus = hipGetLastError();
		if(cudaStatus != hipSuccess)
		{
			throw string(hipGetErrorString(cudaStatus)) + "\n";
		}
		if(!m.sync()) throw m.error();
		if(!m.collect<float>("output", S, v.data)) throw m.error();
	}
	catch(string e)
	{
		cout << "error in gpu_transform: " << e << endl;
		return false;
	}

	return true;
}


bool LLGPU_transform(VMat & v, Mat & transformation_matrix_input)
{
	int S = v.s3;
	Mat transformation_matrix = transformation_matrix_input.clone();
	transformation_matrix = transformation_matrix.inv();
	LCuda_Host_Manager m;
	try
	{
		if(!m.set_default_device()) throw m.error();
		if(!m.new_<float>("input", S, v.data)) throw m.error();
		if(!m.new_<float>("output", S)) throw m.error();
		if(!m.new_<float>("matrix", 16, (float*)transformation_matrix.data)) throw m.error();
		dim3 threadsPerBlock(8, 8, 8);
		dim3 numBlocks(v.s / threadsPerBlock.x, v.s / threadsPerBlock.y, v.s / threadsPerBlock.z);
		volume_transform<<<numBlocks, threadsPerBlock>>>(m.at<float>("input"), m.at<float>("output"), m.at<float>("matrix"), v.s);
		auto cudaStatus = hipGetLastError();
		if(cudaStatus != hipSuccess)
		{
			throw string(hipGetErrorString(cudaStatus)) + "\n";
		}
		if(!m.sync()) throw m.error();
		if(!m.collect<float>("output", S, v.data)) throw m.error();
	}
	catch(string e)
	{
		cout << "error in gpu_transform: " << e << endl;
		return false;
	}

	return true;
}


bool LLGPU_fft3d(VMat & v, VMat & re, VMat & im)
{
	LCuda_Host_Manager m;
	try
	{
		int S = v.s3;
		if(!m.set_default_device()) throw m.error();
		VMatCufftComplex v_ = v;
		//setup the plan
		hipfftHandle plan;
		hipfftPlan3d(&plan, v.s, v.s, v.s, HIPFFT_C2C);
		//allocate gpu data
		if(!m.new_<hipfftComplex>("v_", S, v_.d)) throw m.error();
		//do fft
		hipfftExecC2C(plan, m.at<hipfftComplex>("v_"), m.at<hipfftComplex>("v_"), HIPFFT_FORWARD);
		auto cudaStatus = hipGetLastError();
		if(cudaStatus != hipSuccess)
			throw string("addKernel launch failed: ") + string(hipGetErrorString(cudaStatus)) + "\n";
		if(!m.sync()) throw m.error();
		//copy out data
		if(!m.collect<hipfftComplex>("v_", S, v_.d)) throw m.error();
		re = v_.real();
		im = v_.imag();
		hipfftDestroy(plan);
	}catch(string s)
	{
		cout << "error in fft3d " << s << endl;
		return false;
	}
	return true;
}

bool LLGPU_ifft3d(VMat & re, VMat & im, VMat & out)
{
	LCuda_Host_Manager m;
	try
	{
		int S = re.s3;
		if(!m.set_default_device()) throw m.error();
		VMatCufftComplex re_im(re,im);
		//setup the plan
		hipfftHandle plan;
		hipfftPlan3d(&plan, re.s, re.s, re.s, HIPFFT_C2C);
		//allocate gpu data
		if(!m.new_<hipfftComplex>("re_im", S, re_im.d)) throw m.error();
		//do fft
		hipfftExecC2C(plan, m.at<hipfftComplex>("re_im"), m.at<hipfftComplex>("re_im"), HIPFFT_BACKWARD);
		auto cudaStatus = hipGetLastError();
		if(cudaStatus != hipSuccess)
			throw string("addKernel launch failed: ") + string(hipGetErrorString(cudaStatus)) + "\n";
		if(!m.sync()) throw m.error();
		//copy out data
		if(!m.collect<hipfftComplex>("re_im", S, re_im.d)) throw m.error();
		out = re_im.real();
		hipfftDestroy(plan);
	}catch(string s)
	{
		cout << "error in ifft3d " << s << endl;
		return false;
	}
	return true;
}

bool LLGPU_fft3d_mag_swap_quads(VMat & v, VMat & output)
{
	LCuda_Host_Manager m;
	try
	{
		int S = v.s3;
		output = VMat(v.s);
		//copy over to cuda data structure v_
		VMatCufftComplex v_ = v;
		//make fft plan
		hipfftHandle plan;
		hipfftPlan3d(&plan, v.s, v.s, v.s, HIPFFT_C2C);
		//allocate data on gpu
		if(!m.new_<hipfftComplex>("v_", S, v_.d)) throw m.error();
		if(!m.new_<float>("output", S, output.data)) throw m.error();
		//fft
		hipfftExecC2C(plan, m.at<hipfftComplex>("v_"), m.at<hipfftComplex>("v_"), HIPFFT_FORWARD);
		if(!m.sync()) throw m.error();
		dim3 threadsPerBlock(8, 8, 8);
		int hw = v.s/2;
		dim3 numBlocks(v.s / threadsPerBlock.x, v.s / threadsPerBlock.y, hw / threadsPerBlock.z);
		//swap quadrants and compute magnitude
		gpu_get_magnitude_swapQuads<<<numBlocks, threadsPerBlock>>>(m.at<hipfftComplex>("v_"),
			m.at<float>("output"), v.s);
		auto cudaStatus = hipGetLastError();
		if(cudaStatus != hipSuccess)
		{
			throw string(hipGetErrorString(cudaStatus)) + "\n";
		}
		if(!m.sync()) throw m.error();
		if(!m.collect<float>("output", S, output.data)) throw m.error();
		hipfftDestroy(plan);
	}catch(string s)
	{
		cout << "error occurred in fft3d_mag_swap_quads() -> " << s << endl;
		return false;
	}
	return true;
}


bool LLGPU_log(VMat & v)
{
	LCuda_Host_Manager m;
	try
	{
		int S = v.s3;
		if(!m.set_default_device()) throw m.error();

		if(!m.new_<float>("v", S, v.data)) throw m.error();
		
		//perform element-wise log
		dim3 threadsPerBlock(8, 8, 8);
		dim3 numBlocks(v.s / threadsPerBlock.x, v.s / threadsPerBlock.y, v.s / threadsPerBlock.z);
		log_on_gpu<<<numBlocks, threadsPerBlock>>>(m.at<float>("v"), v.s);
		auto cudaStatus = hipGetLastError();
		if(cudaStatus != hipSuccess)
			throw string(hipGetErrorString(cudaStatus)) + "\n";
		if(!m.sync()) throw m.error();
		if(!m.collect("v", S, v.data)) throw m.error();
	}catch(string e)
	{
		cout << "error in LLGPU_log() -> " << e << endl;
		return false;
	}
	return true;
}

bool LLGPU_log_polar(VMat & v)
{
	LCuda_Host_Manager m;
	try
	{
		int S = v.s3;
		if(!m.set_default_device()) throw m.error();
		if(!m.new_<float>("input", S, v.data)) throw m.error();
		if(!m.new_<float>("output", S)) throw m.error();
		//perform transform
		dim3 threadsPerBlock(8, 8, 8);
		dim3 numBlocks(v.s / threadsPerBlock.x, v.s / threadsPerBlock.y, v.s / threadsPerBlock.z);
		logpolar3d_gpu<<<numBlocks, threadsPerBlock>>>(m.at<float>("input"), m.at<float>("output"), v.s);
		auto cudaStatus = hipGetLastError();
		if(cudaStatus != hipSuccess)
			throw string(hipGetErrorString(cudaStatus)) + "\n";
		if(!m.sync()) throw m.error();
		if(!m.collect("output", S, v.data)) throw m.error();
	}catch(string e)
	{
		cout << "error in LLGPU_log_polar() -> " << e << endl;
		return false;
	}
	return true;
}

bool LLGPU_log_only(VMat & v)
{
	LCuda_Host_Manager m;
	try
	{
		int S = v.s3;
		if(!m.set_default_device()) throw m.error();
		if(!m.new_<float>("input", S, v.data)) throw m.error();
		if(!m.new_<float>("output", S)) throw m.error();
		//perform transform
		dim3 threadsPerBlock(8, 8, 8);
		dim3 numBlocks(v.s / threadsPerBlock.x, v.s / threadsPerBlock.y, v.s / threadsPerBlock.z);
		logonly3d_gpu<<<numBlocks, threadsPerBlock>>>(m.at<float>("input"), m.at<float>("output"), v.s);
		auto cudaStatus = hipGetLastError();
		if(cudaStatus != hipSuccess)
			throw string(hipGetErrorString(cudaStatus)) + "\n";
		if(!m.sync()) throw m.error();
		if(!m.collect("output", S, v.data)) throw m.error();
	}catch(string e)
	{
		cout << "error in logonly3d_gpu() -> " << e << endl;
		return false;
	}
	return true;
}